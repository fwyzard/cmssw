#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <string>
#include <hip/hip_runtime.h>

#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"

__global__
void print(const char * message, size_t length)
{
  //printf("blockIdx.x, threadIdx.x: %d, %d\n", blockIdx.x, threadIdx.x);
  for (size_t i = blockDim.x * blockIdx.x + threadIdx.x; i < length; i += blockDim.x * gridDim.x)
    printf("%c", message[i]);
}

int main(int argc, const char* argv[])
{
  std::string message;
  if (argc == 1) {
    message = "Hello world!";
  } else {
    message = argv[1];
  }

  char * buffer;
  cudaCheck(hipMalloc(& buffer, message.size()));
  cudaCheck(hipMemcpy(buffer, message.data(), message.size(), hipMemcpyDefault));

  print<<<16,1>>>(buffer, message.size());
  cudaCheck(hipDeviceSynchronize());
  cudaCheck(hipGetLastError());
  std::cout << std::endl;

  print<<<4,4>>>(buffer, message.size());
  cudaCheck(hipDeviceSynchronize());
  cudaCheck(hipGetLastError());
  std::cout << std::endl;

  print<<<1,16>>>(buffer, message.size());
  cudaCheck(hipDeviceSynchronize());
  cudaCheck(hipGetLastError());
  std::cout << std::endl;

  cudaCheck(hipFree(buffer));
}
